#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include "mm.h"
using namespace std;

//declare variables for GPU
int *devInputA, *devInputB , *devOut_share , *devOut_nonshare;

void initGPU()
{   
	//allocate memory space(VRAM) on GPU
	hipMalloc(&devInputA, sizeof(int)* SIZE * SIZE  );
	hipMalloc(&devInputB, sizeof(int)* SIZE * SIZE );
	hipMalloc(&devOut_share, sizeof(int)* SIZE *SIZE  );
	hipMalloc(&devOut_nonshare, sizeof(int)* SIZE *SIZE  );
	
	//copy data from DRAM to VRAM
	hipMemcpy(devInputA, MAT_A, sizeof(int)* SIZE * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(devInputB, MAT_B, sizeof(int)* SIZE * SIZE, hipMemcpyHostToDevice);
	
}

void MatrixMul_CPU(int* A, int* B, int* C)
{
	for(int i=0;i<SIZE;i++){
		for(int j=0;j<SIZE;j++){
			for(int k=0;k<SIZE;k++){
				C[i*SIZE+j] += A[i*SIZE+k] * B[k*SIZE+j];
			}
		}
	}
}
__global__ void MatrixMul_GPUnonshared(int* A, int* B, int* C)
{
	int tx = threadIdx.x; int ty = threadIdx.y;
	int bx = blockIdx.x; int by = blockIdx.y;
	
	int Row = by * TILE_SIZE + ty;
	int Col = bx * TILE_SIZE + tx;
	int Pvalue = 0;
	
	for (int k = 0; k < SIZE; ++k)
		Pvalue += A[Row*SIZE+k] * B[k*SIZE+Col];

	__syncthreads();
	
	C[Row*SIZE+Col] = Pvalue;
}

__global__ void MatrixMul_GPUshared(int* A, int* B, int* C)
{
	int tx = threadIdx.x; int ty = threadIdx.y;
	int bx = blockIdx.x; int by = blockIdx.y;
	
	//allocate shared memory
	__shared__ int A_share[TILE_SIZE][TILE_SIZE];
	__shared__ int B_share[TILE_SIZE][TILE_SIZE];
	
	int Row = by * TILE_SIZE + ty;
	int Col = bx * TILE_SIZE + tx;
	int Pvalue = 0;
	
	for (int m = 0; m < SIZE/TILE_SIZE; ++m) {
		//load data from GPU global memory to GPU shared memory
		A_share[ty][tx] = A[Row*SIZE + m*TILE_SIZE + tx];
		B_share[ty][tx] = B[Col + (m*TILE_SIZE + ty)*SIZE];
		__syncthreads();
		
		for (int k = 0; k < TILE_SIZE; ++k)
			Pvalue += A_share[ty][k] * B_share[k][tx];
		__syncthreads();
	}
	__syncthreads();
	C[Row*SIZE+Col] = Pvalue;
}

int main()
{
	int ExecTime_CPU,ExecTime_GPUnonshared, ExecTime_GPUshared;
	timespec time_begin, time_end;     
	init();
	initGPU();
	dim3 threadsPerBlock(TILE_SIZE,TILE_SIZE);
	dim3 numBlocks(SIZE/TILE_SIZE,SIZE/TILE_SIZE);
	
	clock_gettime(CLOCK_REALTIME, &time_begin);
	MatrixMul_CPU(MAT_A,MAT_B,outCPU);
	clock_gettime(CLOCK_REALTIME, &time_end);
	ExecTime_CPU = timespec_diff_ns(time_begin, time_end);
	cout << "ExecTime_CPU is = "  <<  ExecTime_CPU  << "ns" << endl;

	clock_gettime(CLOCK_REALTIME, &time_begin);
	MatrixMul_GPUnonshared<<<numBlocks,threadsPerBlock>>>(devInputA,devInputB,devOut_nonshare); 
	clock_gettime(CLOCK_REALTIME, &time_end);
	ExecTime_GPUnonshared = timespec_diff_ns(time_begin, time_end);
	cout << "ExecTime_GPUnonshared is = "  <<  ExecTime_GPUnonshared  << "ns" << endl;
	
	clock_gettime(CLOCK_REALTIME, &time_begin);
	MatrixMul_GPUshared<<<numBlocks,threadsPerBlock>>>(devInputA,devInputB,devOut_share); 
	clock_gettime(CLOCK_REALTIME, &time_end);
	ExecTime_GPUshared = timespec_diff_ns(time_begin, time_end);
	cout << "ExecTime_GPUshared is = "  <<  ExecTime_GPUshared  << "ns" << endl;

	//copy data from VRAM to DRAM 
	hipMemcpy(outGPU_nonshare, devOut_nonshare , sizeof(int) * SIZE * SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(outGPU_share   , devOut_share    , sizeof(int) * SIZE * SIZE, hipMemcpyDeviceToHost);
	
	hipFree(&devInputA);
	hipFree(&devInputB);
	hipFree(&devOut_share);
	hipFree(&devOut_nonshare);
	
	if(checker())
		cout << "You pass the check" << endl;
	else 
		cout << "There is something wrong" << endl;
	
	return 0;
}

